
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void MatrixAdd(float* d_out, float* d_in1, float* d_in2, int M, int N)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    float f = d_in1[idx];
    float g = d_in2[idx];
    if(blockIdx.x < M && threadIdx.x < N)
    {
        d_out[idx] = f + g;
    }
}

int main(int argc, char* argv[])
{
    FILE *fp;
    if(argc < 2)
    {
        int i =0;
        while(i != 10)
        {
            printf("WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING\n");
            i++;
        }
        printf("ERROR: NO INPUT FILE SELECTED! USE %s <INPUT FILE> \n\n", argv[0]);
        exit(1);
    }
    char* filename = (char*)malloc(strlen(argv[1])*sizeof(char));
    strcpy(filename, argv[1]);
    fp = fopen(filename, "r");
    if(fp == NULL)
    {
        int i = 0;
        while(i != 10)
        {
            printf("WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING\n");
            i++;
        }
        printf("ERROR: FILE COULD NOT BE READ. CANT EVEN TYPE A FILE NAME EH? NERD\n");
        free(filename);
        fclose(fp);
        exit(1);
    }
    int M, N;
    M = N = 0;
    fscanf(fp, "%d %d", &M, &N);
    int MATRIXMEM = (M*N) * sizeof(float);
    float *h_in1;
    h_in1 = (float*)malloc(MATRIXMEM);
    for(int x = 0; x < M; x++)
    {
        for(int y = 0; y < N; y++)
        {
            h_in1[x*N + y] = 0;
        }
    }
    for(int x = 0; x < M; x++)
    {
        for(int y = 0; y < N; y++)
        {
            fscanf(fp, "%f", &h_in1[x*N + y]);
        }
    }
    fscanf(fp, "%d %d", &M, &N);
    float* h_in2;
    h_in2 = (float*)malloc(MATRIXMEM);
    for(int x = 0; x < M; x++)
    {
        for(int y = 0; y < N; y++)
        {
            h_in2[x*N + y] = 0;
        }
    }
    for(int x = 0; x < M; x++)
    {
        for(int y = 0; y < N; y++)
        {
            fscanf(fp, "%f", &h_in2[x*N + y]);
        }
    }
    float* h_out = (float*)malloc(MATRIXMEM);
    float *d_out, *d_in1, *d_in2;
    hipMalloc(&d_in1, MATRIXMEM);
    hipMalloc(&d_in2, MATRIXMEM);
    hipMalloc(&d_out, MATRIXMEM);

    hipMemcpy(d_in1, h_in1, MATRIXMEM, hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, MATRIXMEM, hipMemcpyHostToDevice);

    MatrixAdd<<<M, N>>>(d_out, d_in1, d_in2, M, N);


    hipMemcpy(h_out, d_out, MATRIXMEM, hipMemcpyDeviceToHost);

    for(int x = 0; x < M; x++)
    {
        for(int y = 0; y < N; y++)
        {
            printf("%12f ", h_out[x*N + y]);
        }
        printf("\n");
    }

    hipFree(d_in1);
    hipFree(d_in2);
    hipFree(d_out);
    free(h_in1);
    free(h_in2);
    free(h_out);
    fclose(fp);
    exit(0);
}