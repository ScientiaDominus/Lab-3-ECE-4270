#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void MatrixAdd(float** d_out, float** d_in1, float** d_in2)
{
    int idx = threadIdx.x;
    int f = d_in1[idx];
    int g = d_in2[idx];
    d_out[idx] = f + g;
}

int main(int argc, char* argv[])
{
    FILE *fp;
    if(argc < 2)
    {
        int i =0;
        while(i != 10)
        {
            printf("WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING\n");
            i++;
        }
        printf("ERROR: NO INPUT FILE SELECTED! USE %s <INPUT FILE> \n\n", argv[0]);
        exit(1);
    }
    char* filename = (char*)malloc(strlen(argv[1])*sizeof(char));
    strcpy(filename, argv[1]);
    fp = fopen(filename, "r");
    if(fp == NULL)
    {
        int i = 0;
        while(i != 10)
        {
            printf("WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING WARNING\n");
            i++;
        }
        printf("ERROR: FILE COULD NOT BE READ. CANT EVEN TYPE A FILE NAME EH? NERD\n");
        free(filename);
        fclose(fp);
        exit(1);
    }
    int MATRIX_SIZE = 0;
    fscanf(fp, "%f", &MATRIX_SIZE);
    int MATRIXMEM = (MATRIX_SIZE*MATRIX_SIZE) * sizeof(float);
    float **h_in1;
    h_in1 = (float**)malloc(MATRIXMEM);
    for(int x = 0; x < MATRIX_SIZE; x++)
    {
        for(int y = 0; y < MATRIX_SIZE; y++)
        {
            h_in1[x][y] = 0;
        }
    }
    for(int x = 0; x < MATRIX_SIZE; x++)
    {
        for(int y = 0; y < MATRIX_SIZE; y++)
        {
            fscanf(fp, "%d", &h_in1[x][y]);
        }
    }
    fscanf(fp, "%f", &MATRIX_SIZE);
    float** h_in2;
    h_in2 = (float**)malloc(MATRIXMEM);
    for(int x = 0; x < MATRIX_SIZE; x++)
    {
        for(int y = 0; y < MATRIX_SIZE; y++)
        {
            h_in2[x][y] = 0;
        }
    }
    for(int x = 0; x < MATRIX_SIZE; x++)
    {
        for(int y = 0; y < MATRIX_SIZE; y++)
        {
            fscanf(fp, "%d", &h_in2[x][y]);
        }
    }
    float** h_out = (float*)malloc(MATRIXMEM);
    float** d_out, d_in1, d_in2;
    hipMalloc(&d_in1, MATRIXMEM);
    hipMalloc(&d_in2, MATRIXMEM);
    hipMalloc(&d_out, MATRIXMEM);

    hipMemcpy(d_in1, h_in1, MATRIXMEM, hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, MATRIXMEM, hipMemcpyHostToDevice);

    MatrixAdd<<<1, 10>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<11, 20>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<21, 30>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<31, 40>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<41, 50>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<51, 60>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<61, 70>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<71, 80>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<81, 90>>>(d_out, d_in1, d_in2);
    MatrixAdd<<<91, 100>>>(d_out, d_in1, d_in2);

    hipMemcpy(h_out, d_out, MATRIXMEM, hipMemcpyDeviceToHost);

    for(int x = 0; x < MATRIX_SIZE; x++)
    {
        for(int y = 0; y < MATRIX_SIZE; y++)
        {
            printf("%5f", h_out[x][y]);
        }
        printf("\n");
    }

    hipFree(d_in1);
    hipFree(d_in2);
    hipFree(d_out);
    free(h_in1);
    free(h_in2);
    free(h_out);
    fclose(fp);
    exit(0);
}