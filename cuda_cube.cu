#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void multAdd(float* d_out, float* d_in)
{
    int idx = threadidx.x;
    int f = d_in(idx);
    d_out(idx) = ((2*f) + 1);
}


int main(int argc, int** argv)
{
    int ARRAY_SIZE = 128;
    int ARRAY_MEM = ARRAY_SIZE*sizeof(float);

    float* h_in = malloc(ARRAY_MEM);
    float* h_out = malloc(ARRAY_MEM);

    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        h_in(i) = float(i);
    }

    hipMalloc(&d_in, ARRAY_MEM);
    hipMalloc(&d_out, ARRAY_MEM);

    cudaMemCpy(h_in, d_in, ARRAY_SIZE, cudaMemCpyHostToDevice);
    
    multadd<<<1, 128>>> (d_in, d_out);
    
    cudaMemCpy(d_out, h_out, ARRAY_SIZE, cudaMemCpyDeviceToHost);

    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        printf("%f\n", h_out(i));
    }
    
    hipFree(d_in);
    hipFree(d_out);

    free(h_in);
    free(h_out);

    return 0;
}