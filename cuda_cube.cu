
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void multAdd(float *d_in, float *d_out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int f = d_in[idx];
    d_out[idx] = ((2*f) + 1);
}


int main(int argc, char** argv)
{
    int ARRAY_SIZE = 128;
    int ARRAY_MEM = ARRAY_SIZE*sizeof(float);

    float* h_in = (float*)malloc(ARRAY_MEM);
    float* h_out = (float*)malloc(ARRAY_MEM);
    float *d_out, *d_in;

    for(int i = 0; i < ARRAY_SIZE; i++)
    {
        h_in[i] = i;
    }

    hipMalloc(&d_in, ARRAY_MEM);
    hipMalloc(&d_out, ARRAY_MEM);

    hipMemcpy(d_in, h_in, ARRAY_MEM, hipMemcpyHostToDevice);
    
    multAdd<<<1, 128>>>(d_in, d_out);
    
    hipMemcpy(h_out, d_out, ARRAY_MEM, hipMemcpyDeviceToHost);

    for(int i = 0; i < ARRAY_SIZE; i += 4)
    {
        printf("%10f %10f %10f %10f\n", h_out[i], h_out[i+1], h_out[i+2], h_out[i+3]);
    }
    
    hipFree(d_in);
    hipFree(d_out);

    free(h_in);
    free(h_out);

    return 0;
}